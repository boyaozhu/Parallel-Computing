/* 
 * Solves the Panfilov model using an explicit numerical scheme.
 * Based on code orginally provided by Xing Cai, Simula Research Laboratory 
 * and reimplementation by Scott B. Baden, UCSD
 * 
 * Modified and  restructured by Didem Unat, Koc University
 *
 */
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;


// Utilities
// 

// Timer
// Make successive calls and take a difference to get the elapsed time.
static const double kMicro = 1.0e-6;
int blocksize = 32; 

double getTime()
{
    struct timeval TV;
    struct timezone TZ;

    const int RC = gettimeofday(&TV, &TZ);
    if(RC == -1) {
            cerr << "ERROR: Bad call to gettimeofday" << endl;
            return(-1);
    }

    return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );

}  // end getTime()

// Allocate a 2D array
double *alloc2D(int m,int n){
   double *E;
   int nx=n, ny=m;
   E = (double*)malloc(sizeof(double)*nx*ny);
   assert(E);
   return(E);
}
    
// Reports statistics about the computation
// These values should not vary (except to within roundoff)
// when we use different numbers of  processes to solve the problem
 double stats(double *E, int m, int n, double *_mx){
     double mx = -1;
     double l2norm = 0;
     int i, j;
     for (j=1; j<=m; j++)
       for (i=1; i<=n; i++) {
	   l2norm += E[j*(n+2)+i]*E[j*(n+2)+i];
	   if (E[j*(n+2)+i] > mx)
	       mx = E[j*(n+2)+i];
      }
     *_mx = mx;
     l2norm /= (double) ((m)*(n));
     l2norm = sqrt(l2norm);
     return l2norm;
 }

// External functions
extern "C" {
    void splot(double *E, double T, int niter, int m, int n);
}
void cmdLine(int argc, char *argv[], double& T, int& n, int& px, int& py, int& plot_freq, int& no_comm, int&num_threads);

__global__ void setBoundaryLeft(double* E_prev, const int m, const int n)
{
    int j = blockIdx.x*blockDim.x+threadIdx.x+1; 
    if (j<=m) E_prev[j*(n+2)+0] = E_prev[j*(n+2)+2];
}
__global__ void setBoundaryRight(double* E_prev, const int m, const int n)
{
    int j = blockIdx.x*blockDim.x+threadIdx.x+1; 
    if (j<=m) E_prev[j*(n+2)+n+1] = E_prev[j*(n+2)+n-1];
}
__global__ void setBoundaryUp(double* E_prev, const int m, const int n)
{
    int i = blockIdx.x*blockDim.x+threadIdx.x+1; 
    if (i<=n) E_prev[i] = E_prev[2*(n+2)+i];
}
__global__ void setBoundaryDown(double* E_prev, const int m, const int n)
{
    int i = blockIdx.x*blockDim.x+threadIdx.x+1; 
    if (i<=n) E_prev[(m+1)*(n+2)+i] = E_prev[(m-1)*(n+2)+i];
}

__global__ void solvePDE(double *E, double *E_prev, const double alpha, const int m, const int n)
{
    int j = blockIdx.y*blockDim.y+threadIdx.y+1; 
    int i = blockIdx.x*blockDim.x+threadIdx.x+1;
    
    if ((j<=m) && (i<=n))
    {
        E[j*(n+2)+i] = E_prev[j*(n+2)+i]+alpha*(E_prev[j*(n+2)+i+1]+E_prev[j*(n+2)+i-1]-4*E_prev[j*(n+2)+i]+E_prev[(j+1)*(n+2)+i]+E_prev[(j-1)*(n+2)+i]); 
    }
}

__global__ void solveODE_E(double *E, double *R,  const double kk,
	       const double dt, const double a, const int m, const int n)
{
    int j = blockIdx.y*blockDim.y+threadIdx.y+1; 
    int i = blockIdx.x*blockDim.x+threadIdx.x+1;
    
    if ((j<=m) && (i<=n))
    {
        E[j*(n+2)+i] = E[j*(n+2)+i] -dt*(kk* E[j*(n+2)+i]*(E[j*(n+2)+i] - a)*(E[j*(n+2)+i]-1)+ E[j*(n+2)+i] *R[j*(n+2)+i]);
    }
}

__global__ void solveODE_R(double *E, double *R,  const double kk,
	       const double dt, const double epsilon, const double M1,
           const double  M2, const double b, const int m, const int n)
{
    int j = blockIdx.y*blockDim.y+threadIdx.y+1; 
    int i = blockIdx.x*blockDim.x+threadIdx.x+1;
    
    if ((j<=m) && (i<=n))
    {
        R[j*(n+2)+i] = R[j*(n+2)+i] + dt*(epsilon+M1* R[j*(n+2)+i]/( E[j*(n+2)+i]+M2))*(-R[j*(n+2)+i]-kk* E[j*(n+2)+i]*(E[j*(n+2)+i]-b-1));
    }
}


void simulate (double* E,  double* E_prev,double* R,
	       const double alpha, const int n, const int m, const double kk,
	       const double dt, const double a, const double epsilon,
	       const double M1,const double  M2, const double b)
{

    /* 
     * Copy data from boundary of the computational box 
     * to the padding region, set up for differencing
     * on the boundary of the computational box
     * Using mirror boundaries
     */
      
    setBoundaryLeft<<<ceil(m/64.0),64>>>(E_prev, m, n); 
    setBoundaryRight<<<ceil(m/64.0),64>>>(E_prev, m, n);
    setBoundaryUp<<<ceil(n/64.0),64>>>(E_prev, m, n);
    setBoundaryDown<<<ceil(n/64.0),64>>>(E_prev, m, n);
    
    
    // Solve for the excitation, the PDE
    dim3 DimGrid(ceil((double)n/blocksize), ceil((double)m/blocksize), 1); 
    dim3 DimBlock(blocksize, blocksize, 1); 
    solvePDE<<<DimGrid, DimBlock>>>(E, E_prev, alpha, m, n); 
    
    /* 
     * Solve the ODE, advancing excitation and recovery to the
     *     next timtestep
     */
    solveODE_E<<<DimGrid, DimBlock>>>(E, R, kk, dt, a, m, n); 
    solveODE_R<<<DimGrid, DimBlock>>>(E, R, kk, dt, epsilon, M1, M2, b, m, n); 
    
}

// Main program
int main (int argc, char** argv)
{
  /*
   *  Solution arrays
   *   E is the "Excitation" variable, a voltage
   *   R is the "Recovery" variable
   *   E_prev is the Excitation variable for the previous timestep,
   *      and is used in time integration
   */
  double *E, *R, *E_prev;
  
  // Various constants - these definitions shouldn't change
  const double a=0.1, b=0.1, kk=8.0, M1= 0.07, M2=0.3, epsilon=0.01, d=5e-5;
  
  double T=1000.0;
  int m=200,n=200;
  int plot_freq = 0;
  int px = 1, py = 1;
  int no_comm = 0;
  int num_threads=1; 

  cmdLine( argc, argv, T, n,px, py, plot_freq, no_comm, num_threads);
  m = n;  
  // Allocate contiguous memory for solution arrays
  // The computational box is defined on [1:m+1,1:n+1]
  // We pad the arrays in order to facilitate differencing on the 
  // boundaries of the computation box
  E = alloc2D(m+2,n+2);
  E_prev = alloc2D(m+2,n+2);
  R = alloc2D(m+2,n+2);
  
  int i,j;
  // Initialization
  for (j=1; j<=m; j++)
    for (i=1; i<=n; i++)
      E_prev[j*(n+2)+i] = R[j*(n+2)+i] = 0;
  
  for (j=1; j<=m; j++)
    for (i=n/2+1; i<=n; i++)
      E_prev[j*(n+2)+i] = 1.0;
  
  for (j=m/2+1; j<=m; j++)
    for (i=1; i<=n; i++)
      R[j*(n+2)+i] = 1.0;
  
  double dx = 1.0/n;

  // For time integration, these values shouldn't change 
  double rp= kk*(b+1)*(b+1)/4;
  double dte=(dx*dx)/(d*4+((dx*dx))*(rp+kk));
  double dtr=1/(epsilon+((M1/M2)*rp));
  double dt = (dte<dtr) ? 0.95*dte : 0.95*dtr;
  double alpha = d*dt/(dx*dx);

  cout << "Grid Size       : " << n << endl; 
  cout << "Duration of Sim : " << T << endl; 
  cout << "Time step dt    : " << dt << endl; 
  cout << "Process geometry: " << px << " x " << py << endl;
  if (no_comm)
    cout << "Communication   : DISABLED" << endl;
  
  cout << endl;
  
  //cuda: transfer data to GPU
  double *E_cuda, *R_cuda, *E_prev_cuda;
  int size = (m+2)*(n+2)*sizeof(double);
  hipMalloc((void **) &E_cuda, size);
  hipMemcpy(E_cuda, E, size, hipMemcpyHostToDevice); 
  hipMalloc((void **) &R_cuda, size);
  hipMemcpy(R_cuda, R, size, hipMemcpyHostToDevice); 
  hipMalloc((void **) &E_prev_cuda, size);
  hipMemcpy(E_prev_cuda, E_prev, size, hipMemcpyHostToDevice); 
  
  // Start the timer
  double t0 = getTime();
  
 
  // Simulated time is different from the integer timestep number
  // Simulated time
  double t = 0.0;
  // Integer timestep number
  int niter=0;
  
  while (t<T) {
    
    t += dt;
    niter++;
 
    simulate(E_cuda, E_prev_cuda, R_cuda, alpha, n, m, kk, dt, a, epsilon, M1, M2, b); 
    
    //swap current E with previous E
    double *tmp = E_cuda; E_cuda = E_prev_cuda; E_prev_cuda = tmp;
    
    if (plot_freq){
      int k = (int)(t/plot_freq);
      if ((t - k * plot_freq) < dt){
        hipMemcpy(E, E_cuda, size, hipMemcpyDeviceToHost);  
	    splot(E,t,niter,m+2,n+2);
      }
    }
  }//end of while loop
  
  double time_elapsed = getTime() - t0;

  double Gflops = (double)(niter * (1E-9 * n * n ) * 28.0) / time_elapsed ;
  double BW = (double)(niter * 1E-9 * (n * n * sizeof(double) * 4.0  ))/time_elapsed;

  cout << "Number of Iterations        : " << niter << endl;
  cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
  cout << "Sustained Gflops Rate       : " << Gflops << endl; 
  cout << "Sustained Bandwidth (GB/sec): " << BW << endl << endl; 

  //cuda: transfer data from GPU
  hipMemcpy(E_prev, E_prev_cuda, size, hipMemcpyDeviceToHost); 
  hipMemcpy(E, E_cuda, size, hipMemcpyDeviceToHost); 
  hipMemcpy(R, R_cuda, size, hipMemcpyDeviceToHost); 
  hipFree(E_cuda); hipFree(R_cuda); hipFree(E_prev_cuda); 
  
  double mx;
  double l2norm = stats(E_prev,m,n,&mx);
  cout << "Max: " << mx <<  " L2norm: "<< l2norm << endl;

  if (plot_freq){
    cout << "\n\nEnter any input to close the program and the plot..." << endl;
    getchar();
  }
  
  free (E);
  free (E_prev);
  free (R);
  
  return 0;
}
